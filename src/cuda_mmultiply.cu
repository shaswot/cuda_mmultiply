#include "hip/hip_runtime.h"
// https://github.com/charitha22/workspace/blob/master/cuda/mm/naive_matrix_multiply.cu

#include <istream>
#include <iostream>
#include <fstream>

#include <stddef.h>
#include <typeinfo>
#include <stdexcept>

#include <math.h>
#include <functional>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

#include <xtensor/xarray.hpp>
#include <xtensor/xio.hpp>
#include <xtensor/xview.hpp>
#include <xtensor/xnpy.hpp>
#include <xtensor/xsort.hpp>

#define ROW_TILE_WIDTH 32
#define COL_TILE_WIDTH 32

template<typename T>
__global__
void naive_matrix_multiply(T *A, T *B, T* C, int width, int C_rows, int C_cols)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;   
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // check boundry conditions
  if( row < C_rows && col < C_cols ){
  /*
    // do the multiplication for one row and col
    T value = 0;
    for(int k = 0; k < width; k++){
      value += A[row * width + k] * B[k * C_cols + col];
    }
    // store result
    C[row * C_cols + col] = value;
   */
   C[row * C_cols + col] = row;
   
    
  }
  

}

template<typename T>
void naive_matrix_multiply_cpu(T *A, T *B, T* C, int width, int C_rows, int C_cols){
  for(int i = 0; i < C_rows; i++)
    for(int j = 0; j < C_cols; j++){
      T value = 0.0f;
      for(int k = 0; k < width; k++){
        value += A[i * width + k] * B[k * C_cols + j];
      }
      C[i * C_cols + j] = value;
    }
}

template<typename T>
bool check_equal(T* A1, T* A2, int rows, int cols){
  for(int i = 0; i < rows; i++)
    for(int j = 0; j < cols; j++){
      if(abs(A1[i * cols + j] - A2[i * cols + j]) > 0.00001){
          return false;
      }
    }
  
  return true;
}


int main(void)
{
    
  // load weights from npy files
  
  xt::xarray<float> matrix_X = xt::load_npy<float>("../data/random_matrix.npy");
  xt::xarray<float> matrix_Y = xt::load_npy<float>("../data/random_input_mat.npy");

  std::cout << "matrix_X SHAPE: " << xt::adapt(matrix_X.shape()) << std::endl;
  std::cout << "matrix_Y SHAPE: " << xt::adapt(matrix_Y.shape()) << std::endl;
  
  unsigned long X_rows = matrix_X.shape()[0];
  unsigned long X_cols = matrix_X.shape()[1];
  
  unsigned long Y_rows = matrix_Y.shape()[0];
  unsigned long Y_cols = matrix_Y.shape()[1];
  
  unsigned long Z_rows = X_rows;
  unsigned long Z_cols = Y_cols;
  
  unsigned long X_size = X_rows * X_cols;
  unsigned long Y_size = Y_rows * Y_cols;
  unsigned long Z_size = Z_rows * Z_cols;
  
  // host copies of X,Y,Z
  float *X = new float[X_size];
  float *Y = new float[Y_size]; 
  float *Z = new float[Z_size];
  float *Z_cpu = new float[Z_size];
  
  // auto data()const: Returns a constant pointer to the underlying array serving as element storage. 
  // The pointer is such that range [data(); data() + size()] is always a valid range, even if the container is empty (data() is not is not dereferenceable in that case)
  
  X = matrix_X.data();
  Y = matrix_Y.data();
  
  // device copies of X, Y, Z
  float *d_X, *d_Y, *d_Z;
  
  // Allocate space for device copies of X, Y, Z
  hipMalloc((void **)&d_X, X_size);
  hipMalloc((void **)&d_Y, Y_size);
  hipMalloc((void **)&d_Z, Z_size);
  
  // Copy a & b from the host to the device
  hipMemcpy(d_X, &X, X_size, hipMemcpyHostToDevice);
  hipMemcpy(d_Y, &Y, Y_size, hipMemcpyHostToDevice);
  
  // Matrix Multiplication on GPU
  //dim3 dim_grid(Z_cols/COL_TILE_WIDTH, Z_rows/ROW_TILE_WIDTH, 1);
  //dim3 dim_block(COL_TILE_WIDTH, ROW_TILE_WIDTH, 1);
  
  dim3 dim_grid(1, 1, 1);
  dim3 dim_block(COL_TILE_WIDTH, ROW_TILE_WIDTH, 1);

  naive_matrix_multiply<float><<<dim_grid, dim_block>>>(X, Y, Z, X_cols, Z_rows, Z_cols);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  // Copy result back to the host
  hipMemcpy(&Z, d_Z, Z_size, hipMemcpyDeviceToHost);
  
  xt::xarray<double>::shape_type matrix_Z_shape = {Z_rows, Z_cols};
  xt::xarray<float> matrix_Z = xt::adapt(Z, Z_size, xt::acquire_ownership(), matrix_Z_shape);
  std::cout<<"GPU: matrix_Z"<<std::endl;
  std::cout<<matrix_Z<<std::endl;
  std::cout<<"**********************"<<std::endl;

  // Matrix Multiplication on CPU
  naive_matrix_multiply_cpu<float>(X, Y, Z_cpu, X_cols, Z_rows, Z_cols);
  
  xt::xarray<float> matrix_Z_cpu = xt::adapt(Z_cpu, Z_size, xt::acquire_ownership(), matrix_Z_shape);
  std::cout<<"CPU: matrix_Z"<<std::endl;
  std::cout<<matrix_Z_cpu<<std::endl;
  
  
  if(check_equal<float>(Z, Z_cpu, Z_rows, Z_cols))
    std::cout << "PASS" << std::endl;
  else
    std::cout << "FAIL" << std::endl;
    

  // Free memory
  hipFree(d_X);
  hipFree(d_Y);
  hipFree(d_Z);
  
  return 0; 
}