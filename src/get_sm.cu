#include <stdio.h>

#include <stdlib.h>

#include <hip/hip_runtime.h>

// https://forums.developer.nvidia.com/t/any-way-to-know-on-which-sm-a-thread-is-running/19974/15
/* E.D. Riedijk */

__device__ uint get_smid(void) {
     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}

__global__ void kern(int *sm){
   if (threadIdx.x==0)
      sm[blockIdx.x]=get_smid();
}

int main(){

   int N = 100;

   int *sm, *sm_d;

   sm = (int *) malloc(N*sizeof(*sm));

   hipMalloc((void**)&sm_d,N*sizeof(*sm_d));

   kern<<<N,N>>>( sm_d);

   hipMemcpy(sm, sm_d, N*sizeof(int), hipMemcpyDeviceToHost);

   for (int i=0;i<N;i++)

      printf("%d %d\n",i,sm[i]);

return 0;

}